
#include <hip/hip_runtime.h>
#include <__clang_cuda_builtin_vars.h>
__global__ void convolution_1d_kernel(const float* input, const float* kernel,
                                      float* output, int input_size, int kernel_size) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < input_size - kernel_size + 1) {
        float sum = 0;
        for (int i = 0; i < kernel_size; i++) {
            sum += input[x + i] * kernel[i];
        }
        output[x] = sum;
    }
}
