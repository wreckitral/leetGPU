
#include <hip/hip_runtime.h>
__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int col = blockDim.x * blockIdx.x + threadIdx.x; 
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if(row < M && col < K) {
        float sum = 0;

        for(int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[K * i + col];

        }

        C[row * K + col] = sum;
    }
}