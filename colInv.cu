
#include <hip/hip_runtime.h>
__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < width * height) {
        int startIdx = x * 4;
        image[startIdx] = 255 - image[startIdx];
        image[startIdx + 1] = 255 - image[startIdx + 1];
        image[startIdx + 2] = 255 - image[startIdx + 2];
    }
}
