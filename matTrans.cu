
#include <hip/hip_runtime.h>
__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if(j < cols && i < rows) {
        output[j * rows + i] = input[i * cols + j];
    }
}