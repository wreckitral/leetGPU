
#include <hip/hip_runtime.h>
#include <__clang_cuda_builtin_vars.h>
__global__ void reverse_array(float* input, int N) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    float temp;

    if (x < (N + 1) / 2) {
        temp = input[x];
        input[x] = input[N - 1 - x];
        input[N - 1 - x] = temp;
    }
}

// 0, 1, 2, 3, 4, 5 lets say this is a index of an array
// 5, 4, 3, 2, 1, 0
