
#include <hip/hip_runtime.h>
__global__ void copy_matrix_kernel(const float* A, float* B, int N) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < N * N) {
        B[x] = A[x];
    }
}
