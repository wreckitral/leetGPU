
#include <hip/hip_runtime.h>
__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < N) {
        output[x] = fmaxf(input[x], 0.0f) + 0.01f * fminf(input[x], 0.0f);
    }

}
