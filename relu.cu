
#include <hip/hip_runtime.h>
__global__ void relu_kernel(const float* input, float* output, int N) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < N) {
        output[x] = fmaxf(input[x], 0.0f);
    }
}
