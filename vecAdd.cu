
#include <hip/hip_runtime.h>
__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(col < N) {
        C[col] = A[col] + B[col];
    }

}