
#include <hip/hip_runtime.h>
__device__ unsigned int fnv1a_hash(int input) {
    const unsigned int FNV_PRIME = 16777619;
    const unsigned int OFFSET_BASIS = 2166136261;

    unsigned int hash = OFFSET_BASIS;

    for (int byte_pos = 0; byte_pos < 4; byte_pos++) {
        unsigned char byte = (input >> (byte_pos * 8)) & 0xFF;
        hash = (hash ^ byte) * FNV_PRIME;
    }

    return hash;
}

__global__ void fnv1a_hash_kernel(const int* input, unsigned int* output, int N, int R) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < N) {
        int hash = input[x];
        for(int i = 0; i < R; i++) {
            hash = fnv1a_hash(hash);
        }

        output[x] = hash;
    }
}
